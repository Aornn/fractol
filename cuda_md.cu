#include "hip/hip_runtime.h"
// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 
#include "fract.h"
__global__
void mandelbrot(t_mbrot *mbrot)
{
  int color = 0x01FEDC;
  mbrot->x = 0;

  while(mbrot->x < mbrot->image_x)
  {
    mbrot->y = 0;
    while(mbrot->y < mbrot->image_y)
    {
      mbrot->cr = mbrot->x / mbrot->zoom_x + mbrot->x1;
      mbrot->ci = mbrot->y / mbrot->zoom_y + mbrot->y1;
      mbrot->zr = 0;
      mbrot->zi = 0;
      mbrot->i = 0;
      while (mbrot->zr*mbrot->zr + mbrot->zi*mbrot->zi < 4 && mbrot->i < mbrot->itemax)
      {
        mbrot->tmp = mbrot->zr;
        mbrot->zr = mbrot->zr * mbrot->zr - mbrot->zi * mbrot->zi + mbrot->cr;
        mbrot->zi = 2*mbrot->zi*mbrot->tmp+mbrot->ci;
        mbrot->i ++;
      }
      if (mbrot->i != mbrot->itemax)
      { 
        mlx_put_pixel_image(mbrot->x, mbrot->y, mbrot, (color * mbrot->i + 499 / 10) / mbrot->itemax);
      }
      //else
      //  mlx_put_pixel_image(mbrot->x, mbrot->y, mbrot, color);//(color * 49 + mbrot->x) / mbrot->itemax);
      mbrot->y++;
    }
    mbrot->x++;
  }
}